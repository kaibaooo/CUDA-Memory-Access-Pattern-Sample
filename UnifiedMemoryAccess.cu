#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>
__global__ void vecMultiply(int *arr, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size){
        for(int i = 0;i<100000;i++){
            *(arr + tid) += 10;
        }
    }
}

int main(int argc, char *argv[]){
    // Initialize
        int elementSize = 64;
    int threadsPerBlock = 32;
    int blockSize = (elementSize+threadsPerBlock-1)/threadsPerBlock;
    int *host_input_arr;
    hipMallocManaged((void**)&host_input_arr, sizeof(int) * elementSize);
    for(int i = 0;i<elementSize;i++){
        host_input_arr[i] = i;
    }

    
    vecMultiply<<<blockSize, threadsPerBlock>>>(host_input_arr, elementSize);
    hipDeviceSynchronize();

    for(int i = 0;i<elementSize;i++){
        printf("%d ", host_input_arr[i]);
    }
    printf("\n");
    
    hipFree(host_input_arr);
    
    
    return 0;
}