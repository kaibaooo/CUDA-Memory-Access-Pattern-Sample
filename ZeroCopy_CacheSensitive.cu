#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>
__global__ void vecMultiply(int *arr, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size){
        for(int i = 0;i<100000;i++){
            *(arr + tid) += 10;
        }
    }
}

int main(int argc, char *argv[]){
    // Initialize
        int elementSize = 64;
    int threadsPerBlock = 32;
    int blockSize = (elementSize+threadsPerBlock-1)/threadsPerBlock;
    int *host_input_arr;
    int *device_input_arr;
    hipHostAlloc((void**)&host_input_arr, sizeof(int) * elementSize, hipHostMallocMapped);
    for(int i = 0;i<elementSize;i++){
        host_input_arr[i] = i;
    }
    
    hipHostGetDevicePointer((void **)&device_input_arr,  (void *) host_input_arr , 0);
    vecMultiply<<<blockSize, threadsPerBlock>>>(device_input_arr, elementSize);
    hipDeviceSynchronize();

    for(int i = 0;i<elementSize;i++){
        printf("%d ", device_input_arr[i]);
    }
    printf("\n");
    
    hipFree(device_input_arr);
    
    
    return 0;
}