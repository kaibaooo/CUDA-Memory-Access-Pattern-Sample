#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>
__global__ void vecMultiply(int *arr, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size){
        for(int i = 0;i<100000;i++){
            *(arr + tid) += 10;
        }
    }
}

int main(int argc, char *argv[]){
    // Initialize
    struct timeval start;
    struct timeval end;
    unsigned long diff;
    
    int elementSize = 8192;
    int threadsPerBlock = 32;
    int blockSize = (elementSize+threadsPerBlock-1)/threadsPerBlock;
    int *host_input_arr;
    int *host_output_arr;
    int *device_arr;
    hipHostMalloc((void**)&host_input_arr, sizeof(int) * elementSize, hipHostMallocDefault, hipHostMallocDefault);
    hipHostMalloc((void**)&host_output_arr, sizeof(int) * elementSize, hipHostMallocDefault, hipHostMallocDefault);
    for(int i = 0;i<elementSize;i++){
        host_input_arr[i] = i;
    }

    gettimeofday(&start, NULL);
    hipMalloc((void**)&device_arr, sizeof(int) * elementSize);
    hipMemcpy(device_arr, host_input_arr, sizeof(int) * elementSize, hipMemcpyHostToDevice);
    vecMultiply<<<blockSize, threadsPerBlock>>>(device_arr, elementSize);
    hipMemcpy(host_output_arr, device_arr, sizeof(int) * elementSize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // for(int i = 0;i<elementSize;i++){
    //     printf("%d ", host_output_arr[i]);
    // }
    // printf("\n");
    gettimeofday(&end, NULL);
    hipFree(device_arr);
    hipHostFree(host_input_arr);
    hipHostFree(host_output_arr);
    
    diff = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
    printf("Spend Time is %.2fms\n", diff / 1000.0);
    return 0;
}