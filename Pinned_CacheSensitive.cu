#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>
__global__ void vecMultiply(int *arr, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size){
        for(int i = 0;i<100000;i++){
            *(arr + tid) += 10;
        }
    }
}

int main(int argc, char *argv[]){
    // Initialize
        int elementSize = 64;
    int threadsPerBlock = 32;
    int blockSize = (elementSize+threadsPerBlock-1)/threadsPerBlock;
    int *host_input_arr;
    int *host_output_arr;
    int *device_arr;
    hipHostMalloc((void**)&host_input_arr, sizeof(int) * elementSize, hipHostMallocDefault, hipHostMallocDefault);
    hipHostMalloc((void**)&host_output_arr, sizeof(int) * elementSize, hipHostMallocDefault, hipHostMallocDefault);
    for(int i = 0;i<elementSize;i++){
        host_input_arr[i] = i;
    }

    
    hipMalloc((void**)&device_arr, sizeof(int) * elementSize);
    hipMemcpy(device_arr, host_input_arr, sizeof(int) * elementSize, hipMemcpyHostToDevice);
    vecMultiply<<<blockSize, threadsPerBlock>>>(device_arr, elementSize);
    hipMemcpy(host_output_arr, device_arr, sizeof(int) * elementSize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    for(int i = 0;i<elementSize;i++){
        printf("%d ", host_output_arr[i]);
    }
    printf("\n");
    
    hipFree(device_arr);
    hipHostFree(host_input_arr);
    hipHostFree(host_output_arr);
    
    
    return 0;
}