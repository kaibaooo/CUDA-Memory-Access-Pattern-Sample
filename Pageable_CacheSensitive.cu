#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>
__global__ void vecMultiply(int *arr, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size){
        for(int i = 0;i<100000;i++){
            *(arr + tid) += 10;
        }
    }
}

int main(int argc, char *argv[]){
    // Initialize
    struct timeval start;
    struct timeval end;
    unsigned long diff;
    
    int elementSize = 8192;
    int threadsPerBlock = 32;
    int blockSize = (elementSize+threadsPerBlock-1)/threadsPerBlock;
    int *host_input_arr = (int*)malloc(sizeof(int) * elementSize);
    int *host_output_arr = (int*)malloc(sizeof(int) * elementSize);
    int *device_arr;
    
    for(int i = 0;i<elementSize;i++){
        host_input_arr[i] = i;
    }

    gettimeofday(&start, NULL);
    hipMalloc((void**)&device_arr, sizeof(int) * elementSize);
    hipMemcpy(device_arr, host_input_arr, sizeof(int) * elementSize, hipMemcpyHostToDevice);
    vecMultiply<<<blockSize, threadsPerBlock>>>(device_arr, elementSize);
    hipMemcpy(host_output_arr, device_arr, sizeof(int) * elementSize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    // for(int i = 0;i<elementSize;i++){
    //     printf("%d ", host_output_arr[i]);
    // }
    // printf("\n");
    gettimeofday(&end, NULL);
    hipFree(device_arr);
    free(host_input_arr);
    free(host_output_arr);
    
    diff = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
    printf("Spend Time is %.2fms\n", diff / 1000.0);
    return 0;
}