#include <hip/hip_runtime.h>

#include <stdio.h>
__global__ void vecMultiply(int *arr, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size){
        for(int i = 0;i<100000;i++){
            *(arr + tid) += 10;
        }
    }
}

int main(int argc, char *argv[]){
    // Initialize
    
    int elementSize = 64;
    int threadsPerBlock = 32;
    int blockSize = (elementSize+threadsPerBlock-1)/threadsPerBlock;
    int *host_input_arr = (int*)malloc(sizeof(int) * elementSize);
    int *host_output_arr = (int*)malloc(sizeof(int) * elementSize);
    int *device_arr;
    
    for(int i = 0;i<elementSize;i++){
        host_input_arr[i] = i;
    }

    hipMalloc((void**)&device_arr, sizeof(int) * elementSize);
    hipMemcpy(device_arr, host_input_arr, sizeof(int) * elementSize, hipMemcpyHostToDevice);
    vecMultiply<<<blockSize, threadsPerBlock>>>(device_arr, elementSize);
    hipMemcpy(host_output_arr, device_arr, sizeof(int) * elementSize, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    for(int i = 0;i<elementSize;i++){
        printf("%d ", host_output_arr[i]);
    }
    printf("\n");
    hipFree(device_arr);
    free(host_input_arr);
    free(host_output_arr);
    
    return 0;
}