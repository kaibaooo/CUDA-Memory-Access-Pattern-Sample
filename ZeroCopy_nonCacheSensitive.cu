#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>
__global__ void vecMultiply(int *arr, int size){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<size){
        *(arr + tid) += 10;
    }
}

int main(int argc, char *argv[]){
    // Initialize
    struct timeval start;
    struct timeval end;
    unsigned long diff;
    
    int elementSize = 8192;
    int threadsPerBlock = 32;
    int blockSize = (elementSize+threadsPerBlock-1)/threadsPerBlock;
    int *host_input_arr;
    int *device_input_arr;
    hipHostAlloc((void**)&host_input_arr, sizeof(int) * elementSize, hipHostMallocMapped);
    for(int i = 0;i<elementSize;i++){
        host_input_arr[i] = i;
    }
    gettimeofday(&start, NULL);
    hipHostGetDevicePointer((void **)&device_input_arr,  (void *) host_input_arr , 0);
    vecMultiply<<<blockSize, threadsPerBlock>>>(device_input_arr, elementSize);
    hipDeviceSynchronize();

    // for(int i = 0;i<elementSize;i++){
    //     printf("%d ", device_input_arr[i]);
    // }
    // printf("\n");
    gettimeofday(&end, NULL);
    hipFree(device_input_arr);
    
    diff = 1000000 * (end.tv_sec - start.tv_sec) + end.tv_usec - start.tv_usec;
    printf("Spend Time is %.2fms\n", diff / 1000.0);
    return 0;
}